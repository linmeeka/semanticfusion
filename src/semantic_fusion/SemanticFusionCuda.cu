/*
 * This file is part of SemanticFusion.
 *
 * Copyright (C) 2017 Imperial College London
 * 
 * The use of the code within this file and all code within files that 
 * make up the software that is SemanticFusion is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/semantic-fusion/semantic-fusion-license/> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 */

#include <stdio.h>
#include <assert.h> 

#include <hip/hip_runtime.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool
        abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    } 
}

/*
@ brief
更新概率图和最大概率图
就是论文里的贝叶斯更新

@ param
ids：map中surfel的ids。map->GetSurfelIdsGpu()
ids_width：map->width
ids_height = map->height();
probabilities：分割结果blob，只读
prob_width = probs->width();
prob_height = probs->height();
prob_channels = probs->channels();
map_table：class_pro，所有surfel所有类别的概率，待更新。class_probabilities_gpu_->mutable_gpu_data()，可写
map_max：class_max，最大概率的map，可写。class_max_gpu_->mutable_gpu_data()
map_size：现在class_pro中的surfel数量。class_probabilities_gpu_->width()
*/
__global__ 
void semanticTableUpdate(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probabilities, const int prob_width, const int prob_height, 
                          const int prob_channels,float* map_table,float* map_max,
                          const int map_size)
{
    // 当前处理的index？
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    // New uniqueness code
    // 在16*16的区域内搜索，定义搜索范围
    const int check_patch = 16;
    const int x_min = (x - check_patch) < 0 ? 0 : (x - check_patch);
    const int x_max = (x + check_patch) > 640 ? 640 : (x + check_patch);
    const int y_min = (y - check_patch) < 0 ? 0 : (y - check_patch);
    int surfel_id = tex2D<int>(ids,x,y);
    int first_h, first_w;
    // 在范围内找到第一个id相等的坐标
    for (int h = y_min; h < 480; ++h) {
        for (int w = x_min; w < x_max; ++w) {
            int other_surfel_id = tex2D<int>(ids,w,h);
            if (other_surfel_id == surfel_id) {
                first_h = h;
                first_w = w;
                break;
            }
        }
    }
    // 检查在不在map里面吗
    if (first_h != y || first_w != x) {
        surfel_id = 0;
    }
    if (surfel_id > 0) {
        // x，y是在map里面的坐标，转到图像上的坐标。即找到surfel对应的像素点
        const int prob_x = static_cast<int>((float(x) / ids_width) * prob_width);
        const int prob_y = static_cast<int>((float(y) / ids_height) * prob_height);
        // 到下一个channel要加的offset，也是图像中的像素个数
        const int channel_offset = prob_width * prob_height;
        // 当前第k帧这个像素的概率（第一个类别）
        const float* probability = probabilities + (prob_y * prob_width + prob_x);
        // 前k-1帧对应surfel的概率（第一个类别）
        float* prior_probability = map_table + surfel_id;
        float total = 0.0;
        // 循环所有类别，累积所有类别前k-1帧概率乘第k帧概率的结果，累加得到total，
        // 这时候prior_probability和probability都指向了最后一个类别的位置
        // 这一步是为了算出来公式里的归一化因子z，并算出各类别乘积的结果
        for (int class_id = 0; class_id < prob_channels; ++class_id) {
            // 
            prior_probability[0] *= probability[0];
            total += prior_probability[0];
            probability += channel_offset;
            prior_probability += map_size;
        }
        // Reset the pointers to the beginning again
        // 把他们重置到最开始的位置（第一个类别）
        probability = probabilities + (prob_y * prob_width + prob_x);
        prior_probability = map_table + surfel_id;
        float max_probability = 0.0;
        int max_class = -1;
        float new_total = 0.0; // 没用到
        // 再次循环所有类别，更新概率图
        // 这一步是为了归一化并得到最大值
        for (int class_id = 0; class_id < prob_channels; ++class_id) {
            // Something has gone unexpectedly wrong - reinitialse
            if (total <= 1e-5) {
                // 若总概率值太小，就所有类别均分概率
                prior_probability[0] = 1.0f / prob_channels;
            } else {
                // 除以归一化因子
                prior_probability[0] /= total;
                // 更新最大的概率值和类别
                if (class_id > 0 && prior_probability[0] > max_probability) {
                    max_probability = prior_probability[0];
                    max_class = class_id;
                }
            }
            // 指向下一个类别
            new_total += prior_probability[0];
            probability += channel_offset;
            prior_probability += map_size;
        }
        // 更新最大的概率。 class_map
        map_max[surfel_id] = static_cast<float>(max_class);
        map_max[surfel_id + map_size] = max_probability;
        map_max[surfel_id + map_size + map_size] += 1.0;
    }
}

/*
@param
ids：map中surfel的ids。map->GetSurfelIdsGpu()
ids_width：map->width 显示图像的长宽？
ids_height = map->height();
probabilities：分割结果blob，只读
prob_width = probs->width(); 这个width是整张图的width <w,h,c,n> n是1 ？
prob_height = probs->height(); 网络输出层
prob_channels = probs->channels();
map_table：class_pro，所有surfel所有类别的概率，待更新。class_probabilities_gpu_->mutable_gpu_data()，可写
map_max：class_max，最大概率的map，可写。class_max_gpu_->mutable_gpu_data()
map_size：现在class_pro中的surfel数量。class_probabilities_gpu_->width()
*/
__host__ 
void fuseSemanticProbabilities(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probabilities, const int prob_width, const int prob_height, 
                          const int prob_channels,float* map_table, float* map_max,
                          const int map_size)
{
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32;
    dim3 dimGrid(blocks,blocks); // 32*32 的 grid
    dim3 dimBlock(640/blocks,480/blocks); // 每一个block中分得 (640/32, 480/32)
    semanticTableUpdate<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,probabilities,
        prob_width,prob_height,prob_channels,map_table,map_max,map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__ 
/*
@ biref
看起来这个函数并没有概率累乘？ 只是在更新table，加入新的surfel，删除旧的surfel(不知道有没有删除)

@ param
n：要更新的数量，surfel的数量乘以pro的height。num_to_update = new_prob_width * prob_height
deleted_ids：要删除的surfel id，指针
num_deleted：要删除的surfel数量
current_table_size：当前的surfel数
probability_table：class_pro的blob数据，只读
prob_width：应该是surfel的数量，n？class_probabilities_gpu_->width()
prob_height：类别的数量，c。class_probabilities_gpu_->height()  prob height is the number of classes
new_prob_width：新map里所有surfle的数量。map->GetMapSurfelCount()
new_probability_table：可写的class_pro_buffer。用于存储新的
map_table：class_max,只读
new_map_table：class_max_buffer，可写
*/
void updateTable(int n, const int* deleted_ids, const int num_deleted, const int current_table_size,
                 float const* probability_table, const int prob_width, const int prob_height, 
                 const int new_prob_width, float* new_probability_table, float const * map_table, float* new_map_table)
{
    // 更新的surfel的指针，因为GPU并行？
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        // index： row： classes col：components
        // 属于哪个类别 行数
        // c*n的矩阵
        // 从0开始？
        const int class_id = index / new_prob_width;
        // 属于这个类别的哪个component  列数
        // 从1开始？
        const int component_id = index - (class_id * new_prob_width);
        // 为什么newid用的是prob_width，而前面的id用的是new prob width?
        // 这里是不是old_id更恰当？ 得到的是这个类别这个surfel在原来map里的id
        const int new_id = (class_id * prob_width) + component_id;
        // 不在删除的范围内，即，是一个新的surfel？ 
        // 新的概率值写入buffer
        if (component_id >= num_deleted) {
            // Initialise to prior (prob height is the number of classes)
            // 建一个新节点，各类概率等分
            new_probability_table[new_id] = 1.0f / prob_height;
            // Reset the max class surfel colouring lookup
            // 这个id位置对应的三行，为负表示不存在最大的
            new_map_table[component_id] = -1.0;
            new_map_table[component_id + prob_width] = -1.0;
            new_map_table[component_id + prob_width + prob_width] = 0.0;
        }
        // 若已经存在，则更新 
        // 同一个surfel，在图里的index不一样，但surfel总数不变？？
        else {
            // 原class_pro中的component编号
            int offset = deleted_ids[component_id];
            // 赋值为原来它的概率
            new_probability_table[new_id] = probability_table[(class_id * prob_width) + offset];
            // Also must update our max class mapping
            // 更新class_max
            // 直接赋值
            new_map_table[component_id] = map_table[offset];
            new_map_table[component_id + prob_width] = map_table[prob_width + offset];
            new_map_table[component_id + prob_width + prob_width] = map_table[prob_width + prob_width + offset];
        }
    }
}

/*
@ param
filtered_ids：*int，map里要删除的surfel id，由map->GetDeletedSurfelIdsGpu()获得 // 为什么删除？
num_filtered：map里要删除surfel数量，由map->GetMapSurfelDeletedCount()获得
current_table_size：当前table大小，当前surfel的数量，每次调用这个函数后更新
probability_table：class_pro的gpu数据（BLOB），所有点所有类别的概率图。class_probabilities_gpu_->gpu_data()获得。是一个只读的
prob_width：图里点的数量。class_probabilities_gpu_->width()，(w,h,c,n)的blob，w是1？？
prob_height：类别数。class_probabilities_gpu_->height()，h也是1？？
new_prob_width：新的map里所有surfle的数量。map->GetMapSurfelCount()。
new_probability_table：新的table，写入buffer。class_probabilities_gpu_buffer_->mutable_gpu_data()，mutable意味着可写
map_table：保存每个surfel当前概率最大的类别的table。class_max_gpu_->gpu_data()，只读。
new_map_table：新的table，class_max_gpu_buffer_->mutable_gpu_data()，可写。
*/
__host__ 
void updateProbabilityTable(int* filtered_ids, const int num_filtered, const int current_table_size,
                            float const* probability_table, const int prob_width, const int prob_height, 
                            const int new_prob_width, float* new_probability_table, 
                            float const* map_table, float* new_map_table)
{
    const int threads = 512;
    // 待更新的数量=新的map中点的数量*类别数。即所有点的所有概率都要更新
    const int num_to_update = new_prob_width * prob_height;
    // 每个线程要负责多少个
    const int blocks = (num_to_update + threads - 1) / threads;
    // dim3:三维向量(x,y,z)
    // GPU的某种设定
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateTable<<<dimGrid,dimBlock>>>(num_to_update,filtered_ids,num_filtered,current_table_size,
        probability_table,prob_width,prob_height,new_prob_width,new_probability_table, 
        map_table, new_map_table);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

/*
@ brief
把class_pro中的概率投影到render_map上，render map是和elastic fusion的map一样的，用于最后的可视化
@ param
ids：map里的surfelid 。map->GetSurfelIdsGpu()
ids_width：id_width = map->width()
ids_height：id_height = map->height()
probability_table：class_pro，可写。class_probabilities_gpu_->mutable_gpu_data()
prob_width：n
table_height：c
rendered_probabilities：rendered_class_probabilities_gpu_->mutable_gpu_data()
*/
__global__ 
void renderProbabilityMapKernel(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probability_table, const int prob_width, const int prob_height, 
                          float* rendered_probabilities) 
{
    // 当前处理的surfel
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int surfel_id = tex2D<int>(ids,x,y);
    // surfel的索引
    int projected_probability_offset = y * ids_width + x;
    int probability_table_offset = surfel_id;
    // 对于所有类别
    for (int class_id = 0; class_id < prob_height; ++class_id) {
        // 如果这个点存在
        if (surfel_id > 0) {
            // 把class_pro中该surfel对应类别的概率赋给render_pro
            rendered_probabilities[projected_probability_offset] = probability_table[probability_table_offset];
        } 
        else 
        // 否则，render_pro中该位置的概率是1或0。仅第0个类别为1，其余类别为0。
        {
            rendered_probabilities[projected_probability_offset] = ((class_id == 0) ? 1.0 : 0.0);
        }
        // 指向下一个类别的位置。
        projected_probability_offset += (ids_width * ids_height);
        probability_table_offset += prob_width;
    }
}

/*
class_pro概率图投影到render map上。通过这个接口转给GPU运行。
*/
__host__
void renderProbabilityMap(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probability_table, const int prob_width, const int prob_height, 
                          float* rendered_probabilities) 
{
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32;
    dim3 dimGrid(blocks,blocks);
    dim3 dimBlock(ids_width/blocks,ids_height/blocks);
    renderProbabilityMapKernel<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,probability_table,prob_width,prob_height,rendered_probabilities);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

/*
@ brief
更新最大概率值和类别
*/
__global__ 
void updateMaxClassKernel(const int n, const float* probabilities, const int classes,
                          float* map_max, const int map_size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        // Reset the pointers to the beginning again
        const float* probability = probabilities + index;
        probability += map_size;
        float max_probability = 0.0;
        int max_class = -1;
        for (int class_id = 1; class_id < classes; ++class_id) {
            if (probability[0] > max_probability) {
                max_probability = probability[0];
                max_class = class_id;
            }
            probability += map_size;
        }
        map_max[index] = static_cast<float>(max_class);
        map_max[index + map_size] = max_probability;
    }
}

/*
CRF里调用的，更新最大值
*/
__host__ 
void updateMaxClass(const int n, const float* probabilities, const int classes,
                    float* map_max, const int map_size)
{
    const int threads = 512;
    const int blocks = (n + threads - 1) / threads;
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateMaxClassKernel<<<dimGrid,dimBlock>>>(n,probabilities,classes,map_max,map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}
